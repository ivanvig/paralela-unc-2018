#include "hip/hip_runtime.h"
#include <stdio.h>

// #define LIST_SIZE 1610612736 //6 GB of ints
//#define LIST_SIZE 209715200 //500 MB of ints
// #define LIST_SIZE 1048576 // 1MB of ints
// #define LIST_SIZE 65536
#define LIST_SIZE 16384
#define BLOCK_SIZE 1024
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

__device__
inline void SWAP(int32_t *_a,int32_t *_b){int32_t __aux; __aux = *_a; *_a = *_b; *_b = __aux;}

void odd_even_bubble_sort_global(int32_t * list, int32_t list_size);
int assert_sorted (int * list, int list_size);

__global__
void shared_koronel(int32_t * list, int32_t list_size)
{
  __shared__ int32_t slist[2*BLOCK_SIZE];

  int64_t pos = 2*(blockDim.x * blockIdx.x + threadIdx.x);

  slist[pos] = list[pos];
  slist[pos+1] = list[pos+1];
  for (int64_t i = 0; i<list_size; i++){
    int64_t pos_oddeven = pos + (i&1);
    if (pos_oddeven < list_size-1)
      if(slist[pos_oddeven]>slist[pos_oddeven+1])
        SWAP(&slist[pos_oddeven], &slist[pos_oddeven+1]);
    __syncthreads();
  }
  list[pos] = slist[pos];
  list[pos+1] = slist[pos+1];
}

__global__
void global_koronel(int32_t * list, int32_t list_size)
{
  int64_t pos = 2*(blockDim.x * blockIdx.x + threadIdx.x);

  for (int64_t i = 0; i<list_size; i++){
    int64_t pos_oddeven = pos + (i&1);
    if (pos_oddeven < list_size-1)
      if(list[pos_oddeven]>list[pos_oddeven+1])
        SWAP(&list[pos_oddeven], &list[pos_oddeven+1]);
    __syncthreads();
  }
}

int main (){
  srand(time(NULL));

  int * random_numbers = (int *) malloc(sizeof(int)*LIST_SIZE);

  printf("Generando lista aleatoria de %i elementos\n", LIST_SIZE);
  for (int i = 0; i<LIST_SIZE; i++){
    random_numbers[i] = rand()%20;
    // random_numbers[i] = LIST_SIZE - i;
    // random_numbers[i] = 0;
  }
  // random_numbers[0] = 111;
  // random_numbers[LIST_SIZE-1] = -1;
  int start_print = 0;
  int n_prints = 2048;
  printf("Antes de gpu: Elementos desde %i hasta %i \n", start_print, start_print+n_prints);
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers[i]);
  }
  printf("\n");
  printf("Odd even bubble sort con memoria global \n");
  odd_even_bubble_sort_global(random_numbers, LIST_SIZE);

  printf("Chequeando si la lista esta ordenada... \n");
  int elem;
  if (elem = assert_sorted(random_numbers, LIST_SIZE)) {
    printf("LISTA MAL ORDENADA EN ELEM N %i \n", elem);
    for (int i=((elem-100) > 0)*(elem-100); i < (((elem+100) < LIST_SIZE)*(elem+100) + ((elem+100) >= LIST_SIZE)*LIST_SIZE); i++)
      printf("%i ", random_numbers[i]);
    printf("\n");
  } else
    printf("LISTA BIEN ORDENADA \n");

  printf("Despues de gpu: Elementos desde %i hasta %i\n", start_print, start_print+n_prints);
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers[i]);
  }
  printf("\n");

  return 0;
}

__host__
void odd_even_bubble_sort_global (int32_t * list, int32_t list_size)
{
  int32_t * device_list_ref;

  // dim3 dimGrid ((uint)(LIST_SIZE/(2*BLOCK_SIZE)), 1, 1); //TODO: Usar ceil
  dim3 dimGrid (1, 1, 1); //TODO: Usar ceil
	dim3 dimBlock (BLOCK_SIZE, 1, 1);

  CUDA_CALL(hipMalloc((void **) &device_list_ref, list_size*sizeof(int32_t)));
  CUDA_CALL(hipMemcpy(device_list_ref, list, list_size*sizeof(int32_t), hipMemcpyHostToDevice));

  printf("Llamando al kernel... \n");
  for (int i = 0; i < LIST_SIZE; i++)
    for (int j = 0; j < LIST_SIZE/(2*BLOCK_SIZE); j++) {
      int win_size = 2*BLOCK_SIZE - (2*BLOCK_SIZE - (LIST_SIZE - (i&1))%(2*BLOCK_SIZE))*(((j+1)*2*BLOCK_SIZE + (i&1)) > LIST_SIZE);
      // shared_koronel<<<dimGrid, dimBlock, sizeof(int32_t)*win_size>>>((device_list_ref + 2*BLOCK_SIZE*j + (i&1)), win_size);
      shared_koronel<<<dimGrid, dimBlock>>>((device_list_ref + 2*BLOCK_SIZE*j + (i&1)), win_size);
    }

  CUDA_CALL(hipMemcpy(list, device_list_ref, list_size*sizeof(int32_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipFree(device_list_ref));
}

__host__
int assert_sorted (int * list, int list_size)
{
  for (int i=0; i<list_size-1; i++){
    if (list[i] > list[i+1])
      return i+1;
  }
  return 0;
}