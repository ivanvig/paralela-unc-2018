
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define LIST_SIZE 1610612736 //6 GB of ints
#define BLOCK_SIZE 1024

int * random_numbers_cuda = (int *) malloc(sizeof(int)*LIST_SIZE);

__device__
inline void SWAP(int32_t *_a,int32_t *_b){int32_t __aux; __aux = *_a; *_a = *_b; *_b = __aux;}

int assert_sorted(int * list, int list_size);
void odd_even_bubble_sort(int32_t * list, int32_t list_size);

int main (){
  srand(time(NULL));

  int * random_numbers = (int *) malloc(sizeof(int)*LIST_SIZE);

  for (int i = 0; i<LIST_SIZE; i++){
    random_numbers[i] = rand()%20;
  }

  printf("antes de gpu \n");
  for (int i=0; i< 20; i++){
    printf("%i ", random_numbers[i]);
  }
  odd_even_bubble_sort(random_numbers, LIST_SIZE);

  printf("despues de gpu \n");
  for (int i=0; i< 20; i++){
    printf("%i ", random_numbers_cuda[i]);
  }
  //assert_sorted()

  return 0;
}

__global__
void odd_even_bubble_sort_kernel(int32_t * unsorted_list, int32_t list_size)
{
  int32_t block_id = blockDim.x * blockIdx.x;
  int32_t thread_id = threadIdx.x;

  if (unsorted_list[(block_id+thread_id)*2] > unsorted_list[(block_id+thread_id)*2+1])
    SWAP(&unsorted_list[(block_id+thread_id)*2], &unsorted_list[(block_id+thread_id)*2+1]);

}



__host__
void odd_even_bubble_sort(int32_t * list, int32_t list_size)
{
  int32_t * cuda_dev;
  hipError_t error;
  error = hipMalloc(&cuda_dev, sizeof(int32_t)*list_size);

  hipMemcpy(cuda_dev, list, sizeof(int32_t)*list_size, hipMemcpyHostToDevice);

  /* odd_even_bubble_sort_kernel<<<(LIST_SIZE/(2*BLOCK_SIZE),1,1), (BLOCK_SIZE,1,1)>>>(list, list_size); */
  odd_even_bubble_sort_kernel<<<(LIST_SIZE/(2*BLOCK_SIZE)), BLOCK_SIZE>>>(list, list_size);

  hipMemcpy(random_numbers_cuda, cuda_dev, sizeof(int32_t)*list_size, hipMemcpyDeviceToHost);
}



int assert_sorted (int * list, int list_size)
{
  for (int i=0; i<list_size-1; i++){
    if (list[i] > list[i+1])
      return 1;
  }

  return 0;
}
