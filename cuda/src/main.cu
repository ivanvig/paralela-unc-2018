
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// #define LIST_SIZE 1610612736 //6 GB of ints
#define LIST_SIZE 100 //6 GB of ints
#define BLOCK_SIZE 1024
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

__device__
inline void SWAP(int32_t *_a,int32_t *_b){int32_t __aux; __aux = *_a; *_a = *_b; *_b = __aux;}

int assert_sorted(int * list, int list_size);
void odd_even_bubble_sort(int32_t * list, int32_t list_size);

int main (){
  srand(time(NULL));

  int * random_numbers = (int *) malloc(sizeof(int)*LIST_SIZE);

  for (int i = 0; i<LIST_SIZE; i++){
    random_numbers[i] = rand()%20;
  }

  int start_print = 0;
  int n_prints = 20;
  printf("antes de gpu \n");
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers[i]);
  }
  odd_even_bubble_sort(random_numbers, LIST_SIZE);

  printf("\ndespues de gpu \n");
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers[i]);
  }

  return 0;
}

__global__
void odd_even_bubble_sort_kernel(int32_t * unsorted_list, int32_t list_size)
{
  int32_t block_id = blockDim.x * blockIdx.x;
  int32_t thread_id = threadIdx.x;

  if ((block_id+thread_id)*2<20){
    unsorted_list[(block_id+thread_id)*2]=10;
  }
  // if (unsorted_list[(block_id+thread_id)*2] > unsorted_list[(block_id+thread_id)*2+1])
  //   SWAP(&unsorted_list[(block_id+thread_id)*2], &unsorted_list[(block_id+thread_id)*2+1]);
}



__host__
void odd_even_bubble_sort(int32_t * list, int32_t list_size)
{
  int32_t * cuda_dev;
  dim3 dimGrid ((uint)(LIST_SIZE/(2*BLOCK_SIZE)), 1, 1); //TODO: Usar ceil
	dim3 dimBlock (BLOCK_SIZE, 1, 1);

  CUDA_CALL(hipMalloc(&cuda_dev, sizeof(int32_t)*list_size));

  CUDA_CALL(hipMemcpy(cuda_dev, list, sizeof(int32_t)*list_size, hipMemcpyHostToDevice));

  odd_even_bubble_sort_kernel<<<dimGrid, dimBlock>>>(list, list_size);

  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipMemcpy(list, cuda_dev, sizeof(int32_t)*list_size, hipMemcpyDeviceToHost));
}



int assert_sorted (int * list, int list_size)
{
  for (int i=0; i<list_size-1; i++){
    if (list[i] > list[i+1])
      return 1;
  }

  return 0;
}
