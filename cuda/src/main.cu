#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

// #define LIST_SIZE 1610612736 //6 GB of ints
//#define LIST_SIZE 209715200 //500 MB of ints
// #define LIST_SIZE 1048576 // 1MB of ints
// #define LIST_SIZE 65536
// #define LIST_SIZE 16384
#define LIST_SIZE (16384 + 8192)
// #define LIST_SIZE (8192 + 4096 + 2048)
// #define LIST_SIZE (8192 + 4096 + 1024)
// #define LIST_SIZE (3*(8192 + 4096))
// #define LIST_SIZE 8192
// #define LIST_SIZE 4096
#define BLOCK_SIZE 1024
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

__device__
inline void SWAP(int32_t *_a,int32_t *_b){int32_t __aux; __aux = *_a; *_a = *_b; *_b = __aux;}

void odd_even_bubble_sort_global(int32_t * list, int32_t list_size);
void odd_even_bubble_sort_shared(int32_t * list, int32_t list_size);
int assert_sorted (int * list, int list_size);

// __global__
// void shared_koronel_64(int32_t * list, int32_t list_size, int8_t even)
// {
//   __shared__ int32_t slist[2*BLOCK_SIZE+1];
//   int32_t *win = (list + 2*(blockDim.x * blockIdx.x));
//   int32_t win_size;

//   if (((blockIdx.x+1)*2*blockDim.x) > list_size)
//       win_size = list_size % (2*blockDim.x);
//   else
//       win_size = 2*blockDim.x;

//   if (2*threadIdx.x < win_size - 1) {
//       if (even) {
//           *((int64_t *)slist + threadIdx.x) = *((int64_t *)win + threadIdx.x);
//       } else {
//           if (threadIdx.x == 0){
//               // printf("%d %d\n", even, win);
//               slist[1] = win[0];
//           } else {
//               *((int64_t *)slist + threadIdx.x) = *((int64_t *)(win-1) + threadIdx.x);
//           }
//       }
//   }

//   for (int32_t i = 0; i<win_size; i++){
//     int32_t pos_oddeven = 2*threadIdx.x + (i&1);
//     if (pos_oddeven < win_size - 1)
//       if(slist[pos_oddeven]>slist[pos_oddeven+1])
//         SWAP(&slist[pos_oddeven], &slist[pos_oddeven+1]);
//     __syncthreads();
//   }

//   if (2*threadIdx.x < win_size - 1) {
//       if (even) {
//           *((int64_t *)win + threadIdx.x) = *((int64_t *)slist + threadIdx.x);
//       } else {
//           if (threadIdx.x == 0){
//               win[0] = slist[1];
//           } else {
//               *((int64_t *)(win-1) + threadIdx.x) = *((int64_t *)slist + threadIdx.x);
//           }
//       }
//   }
// }

__global__
void shared_koronel(int32_t * list, int32_t list_size)
{
  // if (threadIdx.x == 0 && list_size < 10000) {
  //   printf("Primer elemento: %d\n", *list);
  // }
  __shared__ int32_t slist[2*BLOCK_SIZE];
  int32_t *win = (list + 2*(blockDim.x * blockIdx.x));
  int32_t win_size = 2*blockDim.x - (2*blockDim.x - list_size%(2*blockDim.x))*(((blockIdx.x+1)*2*blockDim.x) > list_size);

  if (2*threadIdx.x < win_size - 1) {
    slist[2*threadIdx.x] = win[2*threadIdx.x];
    slist[2*threadIdx.x + 1] = win[2*threadIdx.x+1];
    // slist[threadIdx.x] = win[threadIdx.x];
    // slist[threadIdx.x + blockDim.x] = win[threadIdx.x+blockDim.x];
  }

  for (int32_t i = 0; i<win_size; i++){
    int32_t pos_oddeven = 2*threadIdx.x + (i&1);
    if (pos_oddeven < win_size - 1)
      if(slist[pos_oddeven]>slist[pos_oddeven+1])
        SWAP(&slist[pos_oddeven], &slist[pos_oddeven+1]);
    __syncthreads();
  }

  if (2*threadIdx.x < win_size - 1) {
    win[2*threadIdx.x] = slist[2*threadIdx.x];
    win[2*threadIdx.x+1] = slist[2*threadIdx.x+1];
    // win[threadIdx.x] = slist[threadIdx.x];
    // win[threadIdx.x + blockDim.x] = slist[threadIdx.x+blockDim.x];
  }
}

__global__
void global_koronel(int32_t * list, int32_t list_size)
{
  int32_t *win = (list + 2*(blockDim.x * blockIdx.x));
  int32_t win_size = 2*blockDim.x - (2*blockDim.x - list_size%(2*blockDim.x))*(((blockIdx.x+1)*2*blockDim.x) > list_size);
  for (int32_t i = 0; i<win_size; i++){
    int32_t pos_oddeven = 2*threadIdx.x + (i&1);
    if (pos_oddeven < win_size - 1)
      if(win[pos_oddeven]>win[pos_oddeven+1])
        SWAP(&win[pos_oddeven], &win[pos_oddeven+1]);
    __syncthreads();
  }
}

int main (){
  srand(time(NULL));

  int * random_numbers_global = (int *) malloc(sizeof(int)*LIST_SIZE);
  int * random_numbers_shared = (int *) malloc(sizeof(int)*LIST_SIZE);

  printf("Generando lista aleatoria de %i elementos\n", LIST_SIZE);
  for (int i = 0; i<LIST_SIZE; i++){
    // random_numbers_global[i] = rand()%20;
    random_numbers_global[i] = LIST_SIZE - i;
  }

  memcpy(random_numbers_shared, random_numbers_global, sizeof(int)*LIST_SIZE);
  int start_print = 0;
  // int n_prints = 4096;
  int n_prints = LIST_SIZE;
  int elem;

  printf("Lista antes de gpu: Elementos desde %i hasta %i \n", start_print, start_print+n_prints);
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers_global[i]);
  }
  printf("\n");

  //*************************************
  // ODD-EVEN BUBBLE SORT CON GLOBAL MEM
  //*************************************

  printf("Odd even bubble sort con memoria global \n");
  // odd_even_bubble_sort_global(random_numbers_global, LIST_SIZE);

  printf("Despues de gpu (global): Elementos desde %i hasta %i\n", start_print, start_print+n_prints);
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers_global[i]);
  }
  printf("\n");

  printf("Chequeando si la lista con global mem esta ordenada... \n");
  if (elem = assert_sorted(random_numbers_global, LIST_SIZE)) {
    printf("LISTA MAL ORDENADA EN ELEM N %i \n", elem);
    for (int i=((elem-100) > 0)*(elem-100); i < (((elem+100) < LIST_SIZE)*(elem+100) + ((elem+100) >= LIST_SIZE)*LIST_SIZE); i++)
      printf("%i ", random_numbers_global[i]);
    printf("\n");
  } else
    printf("LISTA CON GLOBAL MEM BIEN ORDENADA \n");

  printf("Finalizado sort con memoria global \n");

  //*************************************
  // ODD-EVEN BUBBLE SORT CON SHARED MEM
  //*************************************

  printf("Odd even bubble sort con memoria shared \n");
  odd_even_bubble_sort_shared(random_numbers_shared, LIST_SIZE);

  printf("Despues de gpu (shared): Elementos desde %i hasta %i\n", start_print, start_print+n_prints);
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers_shared[i]);
  }
  printf("\n");

  printf("Chequeando si la lista con shared mem esta ordenada... \n");
  if (elem = assert_sorted(random_numbers_shared, LIST_SIZE)) {
    printf("LISTA MAL ORDENADA EN ELEM N %i \n", elem);
    for (int i=((elem-100) > 0)*(elem-100); i < (((elem+100) < LIST_SIZE)*(elem+100) + ((elem+100) >= LIST_SIZE)*LIST_SIZE); i++)
      printf("%i ", random_numbers_shared[i]);
    printf("\n");
  } else
    printf("LISTA CON SHARED MEM BIEN ORDENADA \n");

  return 0;
}

__host__
void odd_even_bubble_sort_global (int32_t * list, int32_t list_size)
{
  int32_t * device_list_ref;
  hipEvent_t start, stop;
  CUDA_CALL(hipEventCreate(&start));
  CUDA_CALL(hipEventCreate(&stop));

  dim3 dimGrid ((uint)(LIST_SIZE/(2*BLOCK_SIZE)), 1, 1); //TODO: Usar ceil
  // dim3 dimGrid (1, 1, 1); //TODO: Usar ceil
	dim3 dimBlock (BLOCK_SIZE, 1, 1);

  CUDA_CALL(hipMalloc((void **) &device_list_ref, list_size*sizeof(int32_t)));
  CUDA_CALL(hipMemcpy(device_list_ref, list, list_size*sizeof(int32_t), hipMemcpyHostToDevice));

  printf("Llamando al kernel con global memory... \n");
  CUDA_CALL(hipEventRecord(start));
  for (int i = 0; i < LIST_SIZE; i++){
    if (i%(LIST_SIZE/10)==0)
      printf("%d/100...\n", 10*i/(LIST_SIZE/10));

    global_koronel<<<dimGrid, dimBlock>>>((device_list_ref + (i&1)), LIST_SIZE - (i&1));
  }
  CUDA_CALL(hipEventRecord(stop));
  CUDA_CALL(hipEventSynchronize(stop));
  float milliseconds = 0;
  CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Tiempo en kernel de global (ms): %f\n", milliseconds/1000);

  CUDA_CALL(hipMemcpy(list, device_list_ref, list_size*sizeof(int32_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipFree(device_list_ref));
}

__host__
void odd_even_bubble_sort_shared (int32_t * list, int32_t list_size)
{
  int32_t * device_list_ref;
  hipEvent_t start, stop;
  CUDA_CALL(hipEventCreate(&start));
  CUDA_CALL(hipEventCreate(&stop));

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  uint maxnblocks = prop.maxThreadsPerMultiProcessor/BLOCK_SIZE * prop.multiProcessorCount; //TODO: ojo flotante
  uint blocks_needed = (uint) ceil((double)LIST_SIZE/(2*BLOCK_SIZE));

  // dim3 dimGrid ((uint)(LIST_SIZE/(2*BLOCK_SIZE)), 1, 1); //TODO: Usar ceil
  // if (LIST_SIZE/(2*BLOCK_SIZE) > maxnblocks) {
  //     dim3 dimGrid (maxnblocks, 1, 1); //TODO: Usar ceil
  // }else{
  //     dim3 dimGrid ((uint)(LIST_SIZE/(2*BLOCK_SIZE)), 1, 1); //TODO: Usar ceil
  // }
  dim3 dimGridMax (maxnblocks, 1, 1); //TODO: Usar ceil
	dim3 dimBlock (BLOCK_SIZE, 1, 1);

  CUDA_CALL(hipMalloc((void **) &device_list_ref, list_size*sizeof(int32_t)));
  CUDA_CALL(hipMemcpy(device_list_ref, list, list_size*sizeof(int32_t), hipMemcpyHostToDevice));

  printf("Llamando al kernel con shared memory... \n");
  CUDA_CALL(hipEventRecord(start));
  for (int i = 0; i < LIST_SIZE; i++){
    if (i%(LIST_SIZE/10)==0)
      printf("%d/100...\n", 10*i/(LIST_SIZE/10));

    for (int j = 0; j < blocks_needed/maxnblocks; j++) {
        if (i == 0)
          printf(
                 "ADENTRO DEL FOR, list_size_kernel: %d, maxnblocks: %d, blocks_needed:%d\n", 
                 LIST_SIZE - (i&1) - (j*2*BLOCK_SIZE*maxnblocks),
                 maxnblocks, blocks_needed
                 );
        shared_koronel<<<dimGridMax, dimBlock>>>
          (
           ((j*2*BLOCK_SIZE*maxnblocks) + device_list_ref + (i&1)), 
           LIST_SIZE - (i&1) - (j*2*BLOCK_SIZE*maxnblocks)
           );
    }

    if (blocks_needed % maxnblocks) {
        if (i == 0)
            printf("adentro del if, dos: %d\n list_size:%d\n", blocks_needed%maxnblocks, LIST_SIZE - (i&1) - ((blocks_needed/maxnblocks)*maxnblocks*2*BLOCK_SIZE));
        dim3 dimGridMod (blocks_needed % maxnblocks, 1, 1); //TODO: Usar ceil
        shared_koronel<<<dimGridMod, dimBlock>>>(
            ((blocks_needed/maxnblocks)*maxnblocks*2*BLOCK_SIZE + device_list_ref + (i&1)), 
            LIST_SIZE - (i&1) - ((blocks_needed/maxnblocks)*maxnblocks*2*BLOCK_SIZE)
            );
    }

    // shared_koronel_64<<<dimGrid, dimBlock>>>((device_list_ref + (i&1)), LIST_SIZE - (i&1), (~i)&1);
  }
  CUDA_CALL(hipEventRecord(stop));
  CUDA_CALL(hipEventSynchronize(stop));
  float milliseconds = 0;
  CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Tiempo en kernel de shared (ms): %f\n", milliseconds/1000);

  CUDA_CALL(hipMemcpy(list, device_list_ref, list_size*sizeof(int32_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipFree(device_list_ref));
}

__host__
int assert_sorted (int * list, int list_size)
{
  for (int i=0; i<list_size-1; i++){
    if (list[i] > list[i+1])
      return i+1;
  }
  return 0;
}
