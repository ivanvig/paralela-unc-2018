#include "hip/hip_runtime.h"
#include <stdio.h>

// #define LIST_SIZE 1610612736 //6 GB of ints
//#define LIST_SIZE 209715200 //500 MB of ints
#define LIST_SIZE 2048
#define BLOCK_SIZE 1024
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

__device__
inline void SWAP(int32_t *_a,int32_t *_b){int32_t __aux; __aux = *_a; *_a = *_b; *_b = __aux;}

void odd_even_bubble_sort_global(int32_t * list, int32_t list_size);
int assert_sorted (int * list, int list_size);

__global__
void koronel(int32_t * list, int32_t list_size)
{
  int64_t pos = 2*(blockDim.x * blockIdx.x + threadIdx.x);

  for (int64_t i = 0; i<list_size; i++){
    int64_t pos_oddeven = pos + (i&1);
    if (pos_oddeven < list_size-1)
      if(list[pos_oddeven]>list[pos_oddeven+1])
        SWAP(&list[pos_oddeven], &list[pos_oddeven+1]);
    __syncthreads();
  }
}

int main (){
  srand(time(NULL));

  int * random_numbers = (int *) malloc(sizeof(int)*LIST_SIZE);

  printf("Generando lista aleatoria de %i elementos\n", LIST_SIZE);
  for (int i = 0; i<LIST_SIZE; i++){
    random_numbers[i] = rand()%20;
  }
  int start_print = 0;
  int n_prints = 2048;
  printf("Antes de gpu: Elementos desde %i hasta %i \n", start_print, start_print+n_prints);
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers[i]);
  }
  printf("\n");
  printf("Odd even bubble sort con memoria global \n");
  odd_even_bubble_sort_global(random_numbers, LIST_SIZE);

  printf("Chequeando si la lista esta ordenada... \n");
  int elem;
  if (elem = assert_sorted(random_numbers, LIST_SIZE))
    printf("LISTA MAL ORDENADA EN ELEM N %i \n", elem);
  else
    printf("LISTA BIEN ORDENADA \n");

  printf("Despues de gpu: Elementos desde %i hasta %i\n", start_print, start_print+n_prints);
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers[i]);
  }
  printf("\n");

  return 0;
}

__host__
void odd_even_bubble_sort_global (int32_t * list, int32_t list_size)
{
  int32_t * device_list_ref;

  dim3 dimGrid ((uint)(LIST_SIZE/(2*BLOCK_SIZE)), 1, 1); //TODO: Usar ceil
	dim3 dimBlock (BLOCK_SIZE, 1, 1);

  CUDA_CALL(hipMalloc((void **) &device_list_ref, list_size*sizeof(int32_t)));
  CUDA_CALL(hipMemcpy(device_list_ref, list, list_size*sizeof(int32_t), hipMemcpyHostToDevice));

  printf("Llamando al kernel... \n");
  koronel<<<dimGrid, dimBlock>>>(device_list_ref, list_size);

  CUDA_CALL(hipMemcpy(list, device_list_ref, list_size*sizeof(int32_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipFree(device_list_ref));
}

__host__
int assert_sorted (int * list, int list_size)
{
  for (int i=0; i<list_size-1; i++){
    if (list[i] > list[i+1])
      return i+1;
  }
  return 0;
}