#include "hip/hip_runtime.h"
#include <stdio.h>

// #define LIST_SIZE 1610612736 //6 GB of ints
//#define LIST_SIZE 209715200 //500 MB of ints
// #define LIST_SIZE 1048576 // 1MB of ints
// #define LIST_SIZE 65536
#define LIST_SIZE 16384
#define BLOCK_SIZE 1024
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

__device__
inline void SWAP(int32_t *_a,int32_t *_b){int32_t __aux; __aux = *_a; *_a = *_b; *_b = __aux;}

void odd_even_bubble_sort_global(int32_t * list, int32_t list_size);
void odd_even_bubble_sort_shared(int32_t * list, int32_t list_size);
int assert_sorted (int * list, int list_size);

__global__
void shared_koronel(int32_t * list, int32_t list_size)
{
  __shared__ int32_t slist[2*BLOCK_SIZE];

  int64_t pos = 2*(blockDim.x * blockIdx.x + threadIdx.x);

  int win_size = 2*blockDim.x - (2*blockDim.x - list_size%(2*blockDim.x))*(((blockIdx.x+1)*2*blockDim.x) > list_size);

  slist[threadIdx.x] = list[pos];
  slist[threadIdx.x + 1] = list[pos+1];
  for (int64_t i = 0; i<win_size; i++){
    int64_t pos_oddeven = pos + (i&1);
    if (pos_oddeven < win_size-1)
      if(slist[pos_oddeven]>slist[pos_oddeven+1])
        SWAP(&slist[pos_oddeven], &slist[pos_oddeven+1]);
    __syncthreads();
  }
  list[pos] = slist[pos];
  list[pos+1] = slist[pos+1];
}

__global__
void global_koronel(int32_t * list, int32_t list_size)
{
  int64_t pos = 2*(blockDim.x * blockIdx.x + threadIdx.x);

  int win_size = 2*blockDim.x - (2*blockDim.x - list_size%(2*blockDim.x))*(((blockIdx.x+1)*2*blockDim.x) > list_size);
  for (int64_t i = 0; i<win_size; i++){
    int64_t pos_oddeven = pos + (i&1);
    if (pos_oddeven < win_size)
      if(list[pos_oddeven]>list[pos_oddeven+1])
        SWAP(&list[pos_oddeven], &list[pos_oddeven+1]);
    __syncthreads();
  }
}

int main (){
  srand(time(NULL));

  int * random_numbers_global = (int *) malloc(sizeof(int)*LIST_SIZE);
  int * random_numbers_shared = (int *) malloc(sizeof(int)*LIST_SIZE);

  printf("Generando lista aleatoria de %i elementos\n", LIST_SIZE);
  for (int i = 0; i<LIST_SIZE; i++){
    random_numbers_global[i] = rand()%20;
    // random_numbers_global[i] = LIST_SIZE - i;
  }

  memcpy(random_numbers_shared, random_numbers_global, LIST_SIZE);
  int start_print = 0;
  int n_prints = 256;
  int elem;

  printf("Lista antes de gpu: Elementos desde %i hasta %i \n", start_print, start_print+n_prints);
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers_global[i]);
  }
  printf("\n");

  //*************************************
  // ODD-EVEN BUBBLE SORT CON GLOBAL MEM
  //*************************************

  printf("Odd even bubble sort con memoria global \n");
  odd_even_bubble_sort_global(random_numbers_global, LIST_SIZE);

  printf("Despues de gpu (global): Elementos desde %i hasta %i\n", start_print, start_print+n_prints);
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers_global[i]);
  }
  printf("\n");

  printf("Chequeando si la lista con global mem esta ordenada... \n");
  if (elem = assert_sorted(random_numbers_global, LIST_SIZE)) {
    printf("LISTA MAL ORDENADA EN ELEM N %i \n", elem);
    for (int i=((elem-100) > 0)*(elem-100); i < (((elem+100) < LIST_SIZE)*(elem+100) + ((elem+100) >= LIST_SIZE)*LIST_SIZE); i++)
      printf("%i ", random_numbers_global[i]);
    printf("\n");
  } else
    printf("LISTA CON GLOBAL MEM BIEN ORDENADA \n");

  printf("Finalizado sort con memoria global \n");

  //*************************************
  // ODD-EVEN BUBBLE SORT CON SHARED MEM
  //*************************************

  printf("Odd even bubble sort con memoria shared \n");
  odd_even_bubble_sort_shared(random_numbers_shared, LIST_SIZE);

  printf("Despues de gpu (shared): Elementos desde %i hasta %i\n", start_print, start_print+n_prints);
  for (int i=start_print; i< start_print+n_prints; i++){
    printf("%i ", random_numbers_shared[i]);
  }
  printf("\n");

  printf("Chequeando si la lista con shared mem esta ordenada... \n");
  if (elem = assert_sorted(random_numbers_shared, LIST_SIZE)) {
    printf("LISTA MAL ORDENADA EN ELEM N %i \n", elem);
    for (int i=((elem-100) > 0)*(elem-100); i < (((elem+100) < LIST_SIZE)*(elem+100) + ((elem+100) >= LIST_SIZE)*LIST_SIZE); i++)
      printf("%i ", random_numbers_shared[i]);
    printf("\n");
  } else
    printf("LISTA CON SHARED MEM BIEN ORDENADA \n");

  return 0;
}

__host__
void odd_even_bubble_sort_global (int32_t * list, int32_t list_size)
{
  int32_t * device_list_ref;
  hipEvent_t start, stop;
  CUDA_CALL(hipEventCreate(&start));
  CUDA_CALL(hipEventCreate(&stop));

  dim3 dimGrid ((uint)(LIST_SIZE/(2*BLOCK_SIZE)), 1, 1); //TODO: Usar ceil
  // dim3 dimGrid (1, 1, 1); //TODO: Usar ceil
	dim3 dimBlock (BLOCK_SIZE, 1, 1);

  CUDA_CALL(hipMalloc((void **) &device_list_ref, list_size*sizeof(int32_t)));
  CUDA_CALL(hipMemcpy(device_list_ref, list, list_size*sizeof(int32_t), hipMemcpyHostToDevice));

  printf("Llamando al kernel con global memory... \n");
  CUDA_CALL(hipEventRecord(start));
  for (int i = 0; i < LIST_SIZE; i++){
    if (i%(LIST_SIZE/10)==0)
      printf("%d/100...\n", 10*i/(LIST_SIZE/10));

    global_koronel<<<dimGrid, dimBlock>>>((device_list_ref + (i&1)), LIST_SIZE - (i&1));
  }
  CUDA_CALL(hipEventRecord(stop));
  CUDA_CALL(hipEventSynchronize(stop));
  float milliseconds = 0;
  CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Tiempo en kernel de global (ms): %f\n", milliseconds/1000);

  CUDA_CALL(hipMemcpy(list, device_list_ref, list_size*sizeof(int32_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipFree(device_list_ref));
}

__host__
void odd_even_bubble_sort_shared (int32_t * list, int32_t list_size)
{
  int32_t * device_list_ref;
  hipEvent_t start, stop;
  CUDA_CALL(hipEventCreate(&start));
  CUDA_CALL(hipEventCreate(&stop));

  dim3 dimGrid ((uint)(LIST_SIZE/(2*BLOCK_SIZE)), 1, 1); //TODO: Usar ceil
	dim3 dimBlock (BLOCK_SIZE, 1, 1);

  CUDA_CALL(hipMalloc((void **) &device_list_ref, list_size*sizeof(int32_t)));
  CUDA_CALL(hipMemcpy(device_list_ref, list, list_size*sizeof(int32_t), hipMemcpyHostToDevice));

  printf("Llamando al kernel con shared memory... \n");
  CUDA_CALL(hipEventRecord(start));
  for (int i = 0; i < LIST_SIZE; i++){
    if (i%(LIST_SIZE/10)==0)
      printf("%d/100...\n", 10*i/(LIST_SIZE/10));

    shared_koronel<<<dimGrid, dimBlock>>>((device_list_ref + (i&1)), LIST_SIZE - (i&1));
  }
  CUDA_CALL(hipEventRecord(stop));
  CUDA_CALL(hipEventSynchronize(stop));
  float milliseconds = 0;
  CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Tiempo en kernel de shared (ms): %f\n", milliseconds/1000);

  CUDA_CALL(hipMemcpy(list, device_list_ref, list_size*sizeof(int32_t), hipMemcpyDeviceToHost));
  CUDA_CALL(hipFree(device_list_ref));
}

__host__
int assert_sorted (int * list, int list_size)
{
  for (int i=0; i<list_size-1; i++){
    if (list[i] > list[i+1])
      return i+1;
  }
  return 0;
}